#include "hip/hip_runtime.h"
/*
    pcudaSAT: Simulating an efficient solution to SAT with active membranes on the GPU 
    This simulator is published on:
    J.M. Cecilia, J.M. García, G.D. Guerrero, M.A. Martínez-del-Amor, I. Pérez-Hurtado,
    M.J. Pérez-Jiménez. Simulating a P system based efficient solution to SAT by using
    GPUs, Journal of Logic and Algebraic Programming, 79, 6 (2010), 317-325

    pcudaSAT is a subproject of PMCGPU (Parallel simulators for Membrane 
                                       Computing on the GPU)   
 
    Copyright (c) 2010 Miguel Á. Martínez-del-Amor (RGNC, University of Seville)
 		       Ginés D. Guerrero (GACOP, University of Murcia)
    
    This file is part of pcudaSAT.
  
    pcudaSAT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    pcudaSAT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with pcudaSAT.  If not, see <http://www.gnu.org/licenses/>. */

#include <cutil_inline.h>
#include <iostream>
#include <math.h>

#include "object.h"

#include "division_kernel.cu"
#include "evolution_kernel.cu"
#include "checkout_kernel.cu"

#define MAX_BLOCKS_X 32768
using namespace std;

extern "C" bool gpuHybridSolver(int N, int M, int T, Object * cnf) {
    uint * d_cnf, dev;
    hipDeviceProp_t deviceProp;
    bool * d_response, response = false;
    uint numMemb = 1;
    uint maxMemb = (uint) pow(2.0, N);
    dim3 grid;
    uint blocksPerRow, rowsPerGrid;

    /* Initialize GPU */
    char * def_dev = getenv("DEFAULT_DEVICE");
    if (def_dev!=NULL)
	hipSetDevice(dev= atoi(def_dev));
    else
	hipSetDevice(dev = cutGetMaxGflopsDeviceId());
    
    cutilSafeCall(hipGetDeviceProperties(&deviceProp, dev));

    uint maxDeviceMemb = deviceProp.maxGridSize[0] * deviceProp.maxGridSize[1];
    uint deviceGlobalMem = maxMemb * T * sizeof(Object);

    // test conditions
    cutilCondition(maxMemb <= maxDeviceMemb);
    cutilCondition(T <= deviceProp.maxThreadsPerBlock);
    cutilCondition(deviceGlobalMem <= deviceProp.totalGlobalMem);

    // allocate device memory 
    cutilSafeCall(hipMalloc((void**)&d_cnf, deviceGlobalMem));
    cutilSafeCall(hipMalloc((void**)&d_response, sizeof(bool)));

    cutilSafeCall(hipMemcpy(d_cnf, cnf, sizeof(Object) * T, hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_response, &response, sizeof(bool), hipMemcpyHostToDevice));
    
    // create and start timer
    uint timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    grid = dim3(1);

    for (int i=0; i<N; i++) {
        division<<<grid, T>>>(d_cnf, numMemb);
        // check for any errors
        cutilCheckMsg("Kernel execution failed");

        numMemb<<=1;

        // setup execution parameters
        if (numMemb <= MAX_BLOCKS_X) {
            // We can use a 1D Grid
            blocksPerRow = numMemb;
            rowsPerGrid  = 1;
        } else {
            // We need to use a 2D Grid
            blocksPerRow = rowsPerGrid = (uint) sqrt(numMemb);

            while ((blocksPerRow * rowsPerGrid) < numMemb)
                blocksPerRow++;
        }

        grid = dim3(blocksPerRow, rowsPerGrid);

	//cout << "blocksx=" << blocksPerRow << ", y="<<rowsPerGrid<<endl;
        evolution<<<grid, T>>>(d_cnf, numMemb);
        // check for any errors
        cutilCheckMsg("Kernel execution failed");
    }

    checkOut<<<grid, T, sizeof(uint) * T>>>(d_cnf, d_response, M, numMemb);
    // check for any errors
    cutilCheckMsg("Kernel execution failed");

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    cout << endl << "Execution time: " << cutGetTimerValue(timer) << " ms" << endl;
    cutilCheckError(cutDeleteTimer(timer));

    cutilSafeCall(hipMemcpy(&response, d_response, sizeof(bool), hipMemcpyDeviceToHost));

    //printf("\nEL RESULTADO ES: %s\n", (response)?"true":"false");

    cutilSafeCall(hipFree(d_cnf));
    cutilSafeCall(hipFree(d_response));

    hipDeviceReset();

    return response;
}
