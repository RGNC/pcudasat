#include "hip/hip_runtime.h"
/*
    pcudaSAT: Simulating an efficient solution to SAT with active membranes on the GPU 
    This simulator is published on:
    J.M. Cecilia, J.M. García, G.D. Guerrero, M.A. Martínez-del-Amor, I. Pérez-Hurtado,
    M.J. Pérez-Jiménez. Simulating a P system based efficient solution to SAT by using
    GPUs, Journal of Logic and Algebraic Programming, 79, 6 (2010), 317-325

    pcudaSAT is a subproject of PMCGPU (Parallel simulators for Membrane 
                                       Computing on the GPU)   
 
    Copyright (c) 2010 Miguel Á. Martínez-del-Amor (RGNC, University of Seville)
 		       Ginés D. Guerrero (GACOP, University of Murcia)
    
    This file is part of pcudaSAT.
  
    pcudaSAT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    pcudaSAT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with pcudaSAT.  If not, see <http://www.gnu.org/licenses/>. */

#ifndef _EVOLUTION_SOUT_KERNEL_CU_
#define _EVOLUTION_SOUT_KERNEL_CU_

#include "object.cu"

/*
__device__ inline void object(Object & o, char var, short int i, short int j) {
	o=(var)<<24;
    	o=o|((i)<<8);
    	o=o|(j&0xFF);
}

__device__ inline char getVar(Object & o) {
    	return (char)(o>>24);
}

__device__ inline ushort getJ(Object & o) {
    	return (ushort) (o&0xFF);
}

__device__ inline ushort getI(Object &o) {
	return (short int) ((o>>8)&0xFFFF);
} */

__global__ static void evolution_sout_d(const uint numMemb, Object * cnf) {
    	const uint bid = blockIdx.x+gridDim.x*blockIdx.y;      
    	const uint tid = threadIdx.x;
    	const uint blockSize = blockDim.x;
    	uint pivot = numMemb;

    	if (bid >= pivot)
        	return;

    	pivot >>= 1;

    	Object o = cnf[bid*blockSize+tid];
    	char var = getVar(o);
    	ushort j = getJ(o);
	ushort i = getI(o);

	if ((bid&pivot)!=0) { // Charge = '+'
        	if (var == 'x' && j==1) {
                	var='r';
                }
		else if (var != 'r' && var != 0) {
			j--;
			if (var == 'y' && j==0) {
				var=0;
				i=0;
			}
		}
	}
        else {   // Charge = '-'
                if (var == 'y' && j==1) {
                        var='r';
                }
                else if (var != 'r' && var != 0) {
                        j--;
                        if (var == 'x' && j==0) {
                                var=0;
                                i=0;
                        }
                }
        }
	__syncthreads();


	o=setObject(var,i,j);

    	cnf[bid*blockSize+tid] = o;
}

#endif // _EVOLUTION_KERNEL_H_
